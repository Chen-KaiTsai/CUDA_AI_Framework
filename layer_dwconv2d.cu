#include "hip/hip_runtime.h"
#include "framework.cuh"

__global__ void DWConv2D_cout(int inputH, int inputW, int cin, int outputH, int outputW, int cout, int batchSize, int stride, int kSize, int pad, float* X, float* W, float* B, float* Y)
{
    unsigned int x_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_global_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z_global_idx = blockIdx.z * blockDim.z + threadIdx.z;

#ifdef __KERNEL_DEBUG__
    if ((x_global_idx + y_global_idx + z_global_idx) == 0) {
        unsigned int x_global_size = gridDim.x * blockDim.x;
        unsigned int y_global_size = gridDim.y * blockDim.y;
        unsigned int z_global_size = gridDim.z * blockDim.z;
        
        printf("Grid Size : [%u, %u, %u]\n", gridDim.x, gridDim.y, gridDim.z);
        printf("Block Size : [%u, %u, %u]\n", blockDim.x, blockDim.y, blockDim.z);
        printf("Global Size : [%u, %u, %u]\n", x_global_size, y_global_size, z_global_size);
        printf("Input Height : %d\n", inputH);
        printf("Input Width : %d\n", inputW);
        printf("Input Channel : %d\n", cin);
        printf("Output Height : %d\n", outputH);
        printf("Output Width : %d\n", outputW);
        printf("Output Channel : %d\n", cout);
        printf("Batch Size : %d\n", batchSize);
        printf("Stride : %d\n", stride);
        printf("Kernel Size : %d\n", kSize);
        printf("Padding : %d\n", pad);
    }
#endif

    int width  = x_global_idx;
    int height = y_global_idx;
    int cOut   = z_global_idx;

    int wSubSize = kSize * kSize;
    int xOneBatchSize = inputH * inputW * cin;
    int xMapSize = inputH * inputW;
    int yOneBatchSize = outputH * outputW * cout;
    int yMapSize = outputH * outputW;

    float  sum;
    int indexW;
	for(int N = 0; N < batchSize; ++N)
	{
		sum    = 0;
		indexW = 0;
		for(int kh = 0; kh < kSize; ++kh)
		{
			for(int kw = 0; kw < kSize; ++kw, ++indexW)
			{
				int hp = height * stride + kh - pad;
				int wp = width  * stride + kw - pad;
				if(hp >= 0 && wp >=0 && hp < inputH && wp < inputW)
					sum += W[cOut * wSubSize + indexW] * X[N * xOneBatchSize + cOut * xMapSize + hp * inputW + wp];
			}
		}
        if(B != nullptr)
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum + B[cOut];
        else
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum;
	}
}


__global__ void DWConv2D_shared(int inputH, int inputW, int cin, int outputH, int outputW, int cout, int batchSize, int stride, int kSize, int pad, float* X, float* W, float* B, float* Y)
{
    unsigned int x_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_global_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z_global_idx = blockIdx.z * blockDim.z + threadIdx.z;

#ifdef __KERNEL_DEBUG__
    if ((x_global_idx + y_global_idx + z_global_idx) == 0) {
        unsigned int x_global_size = gridDim.x * blockDim.x;
        unsigned int y_global_size = gridDim.y * blockDim.y;
        unsigned int z_global_size = gridDim.z * blockDim.z;
        
        printf("Grid Size : [%u, %u, %u]\n", gridDim.x, gridDim.y, gridDim.z);
        printf("Block Size : [%u, %u, %u]\n", blockDim.x, blockDim.y, blockDim.z);
        printf("Global Size : [%u, %u, %u]\n", x_global_size, y_global_size, z_global_size);
        printf("Input Height : %d\n", inputH);
        printf("Input Width : %d\n", inputW);
        printf("Input Channel : %d\n", cin);
        printf("Output Height : %d\n", outputH);
        printf("Output Width : %d\n", outputW);
        printf("Output Channel : %d\n", cout);
        printf("Batch Size : %d\n", batchSize);
        printf("Stride : %d\n", stride);
        printf("Kernel Size : %d\n", kSize);
        printf("Padding : %d\n", pad);
    }
#endif

    int width  = x_global_idx;
    int height = y_global_idx;
    int cOut   = z_global_idx;

    int wSubSize = kSize * kSize;
    int xOneBatchSize = inputH * inputW * cin;
    int xMapSize = inputH * inputW;
    int yOneBatchSize = outputH * outputW * cout;
    int yMapSize = outputH * outputW;

    extern __shared__ float sharedWeight[];

    // Setup shared memory
    int indexW = 0;
#ifdef DEBUG_OLD_SHARE
    for (; indexW < wSubSize; ++indexW)
        sharedWeight[threadIdx.z * wSubSize + indexW] = W[cOut * wSubSize + indexW];
#endif

    int sharedNum = wSubSize * blockDim.z;
    int threadLocalIdx = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    int threadLocalDim = blockDim.x * blockDim.y * blockDim.z;

    /*
    for (int i = 0; i <= sharedNum / threadLocalDim; ++i) {
        if ((i * threadLocalDim + threadLocalIdx) < sharedNum)
            sharedWeight[i * threadLocalDim + threadLocalIdx] = W[(sharedNum * blockIdx.z) + (i * threadLocalDim) + threadLocalIdx];
    }
    */

    for (int i = 0; i < sharedNum; i += threadLocalDim) {
        if ((i + threadLocalIdx) < sharedNum)
            sharedWeight[i + threadLocalIdx] = W[(sharedNum * blockIdx.z) + i + threadLocalIdx];
    }

    __syncthreads();

    float sum;
	for(int N = 0; N < batchSize; ++N)
	{
		sum    = 0;
		indexW = 0;
		for(int kh = 0; kh < kSize; ++kh)
		{
			for(int kw = 0; kw < kSize; ++kw, ++indexW)
			{
				int hp = height * stride + kh - pad;
				int wp = width  * stride + kw - pad;
				if(hp >= 0 && wp >=0 && hp < inputH && wp < inputW)
					sum += sharedWeight[threadIdx.z * wSubSize + indexW] * X[N * xOneBatchSize + cOut * xMapSize + hp * inputW + wp];
			}
		}
        if(B != nullptr)
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum + B[cOut];
        else
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum;
	}
}


framework::dwconv2d::dwconv2d(layer* p, int ks, int stride, int pad, bool use_bias, uint8_t version, dim3 blockSize)
{
    name = "DW-Conv2D";
    useBias = use_bias;
    padSize = pad;
    this->stride = stride>2?2:stride;
    kSize = ks;
    prvLayer = p;
    prvShape = prvLayer->getShape();
    shape.N = prvShape.N;
    shape.C = prvShape.C;
    shape.H = (prvShape.H - kSize + 2 * padSize) / stride + 1;
    shape.W = (prvShape.W - kSize + 2 * padSize) / stride + 1;
    memSize = shape.N * shape.C * shape.H * shape.W * sizeof(float);
    
    // Select compute version (NoShared : 0, WithShared : 1)
    this->version = version;

    hipError_t error_id;
    
    error_id = hipMalloc(&gMEM, memSize);
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

    error_id = hipDeviceSynchronize();
    if (error_id != hipSuccess) {
        printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    weightSize = shape.C * kSize * kSize;
    biasSize = shape.C;
    setNParam(weightSize + int(useBias) * biasSize);
    outputSize = shape.N * shape.C * shape.H * shape.W * sizeof(float);

    dimBlock.x = blockSize.x;
    dimBlock.y = blockSize.y;
    dimBlock.z = blockSize.z;

    dimGrid.x = (shape.W + blockSize.x - 1) / blockSize.x;
    dimGrid.y = (shape.H + blockSize.y - 1) / blockSize.y;
    dimGrid.z = (shape.C + blockSize.z - 1) / blockSize.z;

    jobSize.x = shape.W;
    jobSize.y = shape.H;
    jobSize.z = shape.C;
#ifdef __DEBUG__
    printf("%s\nGlobal Work Size [%u, %u, %u]\nGPU Memory Size : %ld\n", name.c_str(), (dimGrid.x * dimBlock.x), (dimGrid.y * dimBlock.y), (dimGrid.z * dimBlock.z), memSize);
    printf("\nBlockDim : [%u, %u, %u]\nGridDim : [%u, %u, %u]\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
#endif
}

/**
 * @brief Launch Kernel with Hyperparameter set in constructor.
 * 
 */
void framework::dwconv2d::run()
{
    hipError_t error_id;
    float* prvMEM = prvLayer->getGPUMem();

    switch (version) {
        case 0:
	        DWConv2D_cout<<<dimGrid, dimBlock>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
            break;
        case 1:
            int sharedMem = kSize * kSize * dimBlock.z * sizeof(float);
#ifdef __DEBUG__
            printf("%s Shared Memory Size : %d\n", name.c_str(), sharedMem);
#endif
            DWConv2D_shared<<<dimGrid, dimBlock, sharedMem>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
            break;
    }
	hipDeviceSynchronize();
    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
		printf("Error: %d\n%s\n\n", static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
}

/**
 * @brief Set layer weight & bias parameter
 * 
 * @param buffer CPU buffer to initial weight & bias parameter
 */
size_t framework::dwconv2d::setParam(float* buffer)
{
  	hipError_t error_id;

    error_id = hipMalloc(&wMEM, weightSize * sizeof(float));
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipMemcpy(wMEM, buffer, weightSize * sizeof(float), hipMemcpyHostToDevice);
	if (error_id != hipSuccess) {
		printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipDeviceSynchronize();
	if (error_id != hipSuccess) {
		printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
    
    if (useBias) {
        error_id = hipMalloc(&bMEM, biasSize * sizeof(float));
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipMemcpy(bMEM, (buffer + weightSize), biasSize * sizeof(float), hipMemcpyHostToDevice);
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipDeviceSynchronize();
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }
    }
    return weightSize + int(useBias) * biasSize;
}
