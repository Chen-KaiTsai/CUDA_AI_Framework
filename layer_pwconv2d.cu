#include "hip/hip_runtime.h"
#include "framework.cuh"

__global__ void PWConv2D_cout(int inputH, int inputW, int cin, int outputH, int outputW, int cout, int batchSize, int stride, int kSize, int pad, float* X, float* W, float* B, float* Y)
{
    unsigned int x_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_global_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z_global_idx = blockIdx.z * blockDim.z + threadIdx.z;

#ifdef __KERNEL_DEBUG__
    if ((x_global_idx + y_global_idx + z_global_idx) == 0) {
        unsigned int x_global_size = gridDim.x * blockDim.x;
        unsigned int y_global_size = gridDim.y * blockDim.y;
        unsigned int z_global_size = gridDim.z * blockDim.z;
        
        printf("Grid Size : [%u, %u, %u]\n", gridDim.x, gridDim.y, gridDim.z);
        printf("Block Size : [%u, %u, %u]\n", blockDim.x, blockDim.y, blockDim.z);
        printf("Global Size : [%u, %u, %u]\n", x_global_size, y_global_size, z_global_size);
        printf("Input Height : %d\n", inputH);
        printf("Input Width : %d\n", inputW);
        printf("Input Channel : %d\n", cin);
        printf("Output Height : %d\n", outputH);
        printf("Output Width : %d\n", outputW);
        printf("Output Channel : %d\n", cout);
        printf("Batch Size : %d\n", batchSize);
        printf("Stride : %d\n", stride);
        printf("Kernel Size : %d\n", kSize);
        printf("Padding %d\n", pad);
    }
#endif

    int width  = x_global_idx;
    int height = y_global_idx;
    int cOut   = z_global_idx;

    int xOneBatchSize = inputH * inputW * cin;
    int xMapSize = inputH * inputW;
    int yOneBatchSize = outputH * outputW * cout;
    int yMapSize = outputH * outputW;

    float sum;
    for (int N = 0; N < batchSize; ++N)
    {
        sum = 0;
        for (int cIn = 0; cIn < cin; ++cIn)
            sum += W[cOut * cin + cIn] * X[N * xOneBatchSize + cIn * xMapSize + height * inputW + width];
        
        if (height < outputH && width < outputW)
        {
            if(B != nullptr)
                Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum + B[cOut];
            else
                Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum;
        }
    }
}


__global__ void PWConv2D_shared(int inputH, int inputW, int cin, int outputH, int outputW, int cout, int batchSize, int stride, int kSize, int pad, float* X, float* W, float* B, float* Y)
{
    unsigned int x_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_global_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z_global_idx = blockIdx.z * blockDim.z + threadIdx.z;

#ifdef __KERNEL_DEBUG__
    if ((x_global_idx + y_global_idx + z_global_idx) == 0) {
        unsigned int x_global_size = gridDim.x * blockDim.x;
        unsigned int y_global_size = gridDim.y * blockDim.y;
        unsigned int z_global_size = gridDim.z * blockDim.z;
        
        printf("Grid Size : [%u, %u, %u]\n", gridDim.x, gridDim.y, gridDim.z);
        printf("Block Size : [%u, %u, %u]\n", blockDim.x, blockDim.y, blockDim.z);
        printf("Global Size : [%u, %u, %u]\n", x_global_size, y_global_size, z_global_size);
        printf("Input Height : %d\n", inputH);
        printf("Input Width : %d\n", inputW);
        printf("Input Channel : %d\n", cin);
        printf("Output Height : %d\n", outputH);
        printf("Output Width : %d\n", outputW);
        printf("Output Channel : %d\n", cout);
        printf("Batch Size : %d\n", batchSize);
        printf("Stride : %d\n", stride);
        printf("Kernel Size : %d\n", kSize);
        printf("Padding %d\n", pad);
    }
#endif

    int width  = x_global_idx;
    int height = y_global_idx;
    int cOut   = z_global_idx;

    int xOneBatchSize = inputH * inputW * cin;
    int xMapSize = inputH * inputW;
    int yOneBatchSize = outputH * outputW * cout;
    int yMapSize = outputH * outputW;

    // Setup shared memory
    extern __shared__ float sharedWeight[];

#ifdef DEBUG_OLD_SHARE
    for (int cIn = 0; cIn < cin; ++cIn)
        sharedWeight[threadIdx.z * cin + cIn] = W[cOut * cin + cIn];
#endif

    int sharedNum = cin * blockDim.z;
    int threadLocalIdx = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    int threadLocalDim = blockDim.x * blockDim.y * blockDim.z;

    /*
    for (int i = 0; i <= sharedNum / threadLocalDim; ++i) {
        if ((i * threadLocalDim + threadLocalIdx) < sharedNum)
            sharedWeight[i * threadLocalDim + threadLocalIdx] = W[(sharedNum * blockIdx.z) + (i * threadLocalDim) + threadLocalIdx];
    }
    */

    for (int i = 0; i < sharedNum; i += threadLocalDim) {
        if ((i + threadLocalIdx) < sharedNum)
            sharedWeight[i + threadLocalIdx] = W[(sharedNum * blockIdx.z) + i + threadLocalIdx];
    }

    __syncthreads();

    float sum;
    for (int N = 0; N < batchSize; ++N)
    {
        sum = 0;
        for (int cIn = 0; cIn < cin; ++cIn)
            sum += sharedWeight[threadIdx.z * cin + cIn] * X[N * xOneBatchSize + cIn * xMapSize + height * inputW + width];
        
        if (height < outputH && width < outputW)
        {
            if(B != nullptr)
                Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum + B[cOut];
            else
                Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum;
        }
    }
}


framework::pwconv2d::pwconv2d(layer* p, int cout, bool use_bias, uint8_t version, dim3 blockSize)
{
    name = "PW-Conv2D";
    useBias = use_bias;
    padSize = 0;
    this->stride = 1;
    kSize = 1;
    prvLayer = p;
    prvShape = prvLayer->getShape();
    shape.N = prvShape.N;
    shape.C = cout;
    shape.H = (prvShape.H - kSize + 2 * padSize) / stride + 1;
    shape.W = (prvShape.W - kSize + 2 * padSize) / stride + 1;
    memSize = shape.N * shape.C * shape.H * shape.W * sizeof(float);

    // Select compute version (NoShared : 0, WithShared : 1)
    this->version = version;

    hipError_t error_id;

    error_id = hipMalloc(&gMEM, memSize);
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

    error_id = hipDeviceSynchronize();
    if (error_id != hipSuccess) {
        printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    weightSize = shape.C * prvShape.C;
    biasSize = shape.C;
    setNParam(weightSize + int(useBias) * biasSize);
    outputSize = shape.N * shape.C * shape.H * shape.W * sizeof(float);

    dimBlock.x = blockSize.x;
    dimBlock.y = blockSize.y;
    dimBlock.z = blockSize.z;

    dimGrid.x = (shape.W + blockSize.x - 1) / blockSize.x;
    dimGrid.y = (shape.H + blockSize.y - 1) / blockSize.y;
    dimGrid.z = (shape.C + blockSize.z - 1) / blockSize.z;

    jobSize.x = shape.W;
    jobSize.y = shape.H;
    jobSize.z = shape.C;
#ifdef __DEBUG__
    printf("%s\nGlobal Work Size [%u, %u, %u]\nGPU Memory Size : %ld\n", name.c_str(), (dimGrid.x * dimBlock.x), (dimGrid.y * dimBlock.y), (dimGrid.z * dimBlock.z), memSize);
    printf("\nBlockDim : [%u, %u, %u]\nGridDim : [%u, %u, %u]\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
#endif
}

#ifdef USE_CUDNN_CONVOLUTION
void framework::pwconv2d::run()
{
    hipdnnHandle_t handle;
    hipdnnCreate(&handle);

    hipdnnDataType_t dtype = HIPDNN_DATA_FLOAT;
    hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;
    
    hipdnnTensorDescriptor_t prvMEM_desc;
    hipdnnTensorDescriptor_t gMEM_desc;
    hipdnnCreateTensorDescriptor(&prvMEM_desc);
    hipdnnCreateTensorDescriptor(&gMEM_desc);
    hipdnnSetTensor4dDescriptor(prvMEM_desc, format, dtype, prvShape.N, prvShape.C, prvShape.H, prvShape.W);
    hipdnnSetTensor4dDescriptor(gMEM_desc, format, dtype, shape.N, shape.C, shape.H, shape.W);

    hipdnnFilterDescriptor_t W_desc;
    hipdnnCreateFilterDescriptor(&W_desc);
    hipdnnSetFilter4dDescriptor(W_desc, dtype, format, shape.C, prvShape.C, kSize, kSize);

    hipdnnConvolutionDescriptor_t pwconv2d_desc;
    hipdnnConvolutionMode_t mode = HIPDNN_CROSS_CORRELATION;

    hipdnnCreateConvolutionDescriptor(&pwconv2d_desc);
    hipdnnSetConvolution2dDescriptor(pwconv2d_desc, padSize, padSize, stride, stride, 1, 1, mode, dtype);

    hipdnnConvolutionFwdAlgo_t algo;
#if CUDNN_MAJOR == 8
    hipdnnConvolutionFwdAlgoPerf_t algos[HIPDNN_CONVOLUTION_FWD_ALGO_COUNT];
    cudnnGetConvolutionForwardAlgorithm_v7(handle, prvMEM_desc, W_desc, pwconv2d_desc, gMEM_desc, HIPDNN_CONVOLUTION_FWD_ALGO_COUNT, nullptr, algos);
    algo = algos[0].algo;
#else
    hipdnnGetConvolutionForwardAlgorithm(handle, prvMEM_desc, W_desc, pwconv2d_desc, gMEM_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, nullptr, algo)
#endif
    size_t workspaceBytes = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(handle, prvMEM_desc, W_desc, pwconv2d_desc, gMEM_desc, algo, &workspaceBytes);
#ifdef __DEBUG__
    printf("CUDNN info from %s\nWorkspace size: %zuMB\n", name.c_str(), (workspaceBytes / 1048576));
#endif
    hipError_t error_id;
    void* workspaceMEM = nullptr;
    error_id = hipMalloc(&workspaceMEM, workspaceBytes);
    if (error_id != hipSuccess) {
        printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
    }

    const float alpha = 1.0f, beta = 0.0f;
    float* prvMEM = prvLayer->getGPUMem();

    if (useBias) {
#ifdef __DEBUG__
        printf("Using bias has not implemented yet. Fall back to non cuDNN implementation\n");
#endif
        int sharedMem = kSize * kSize * prvShape.C * dimBlock.z * sizeof(float);
        PWConv2D_shared<<<dimGrid, dimBlock, sharedMem>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
    }
    else {
        hipdnnConvolutionForward(handle, &alpha, prvMEM_desc, prvMEM, W_desc, wMEM, pwconv2d_desc, algo, workspaceMEM, workspaceBytes, &beta, gMEM_desc, gMEM);
    }

    hipFree(workspaceMEM);
    hipdnnDestroy(handle);
}
#else
/**
 * @brief Launch Kernel with Hyperparameter set in constructor.
 * 
 */
void framework::pwconv2d::run()
{
    hipError_t error_id;
    float* prvMEM = prvLayer->getGPUMem();

    switch (version) {
        case 0:
	        PWConv2D_cout<<<dimGrid, dimBlock>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
            break;
        case 1:
            int sharedMem = prvShape.C * dimBlock.z * sizeof(float);
#ifdef __DEBUG__
            printf("%s Shared Memory Size : %d\n", name.c_str(), sharedMem);
#endif
            PWConv2D_shared<<<dimGrid, dimBlock, sharedMem>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
            break;
    }
	hipDeviceSynchronize();
    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
		printf("Error: pw %d\n%s\n\n", static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
}
#endif

/**
 * @brief Set layer weight & bias parameter
 * 
 * @param buffer CPU buffer to initial weight & bias parameter
 */
size_t framework::pwconv2d::setParam(float* buffer)
{
  	hipError_t error_id;

    error_id = hipMalloc(&wMEM, weightSize * sizeof(float));
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipMemcpy(wMEM, buffer, weightSize * sizeof(float), hipMemcpyHostToDevice);
	if (error_id != hipSuccess) {
		printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipDeviceSynchronize();
	if (error_id != hipSuccess) {
		printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
    
    if (useBias) {
        error_id = hipMalloc(&bMEM, biasSize * sizeof(float));
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipMemcpy(bMEM, (buffer + weightSize), biasSize * sizeof(float), hipMemcpyHostToDevice);
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipDeviceSynchronize();
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }
    }
    return weightSize + int(useBias) * biasSize;
}
