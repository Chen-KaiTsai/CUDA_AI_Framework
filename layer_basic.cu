#include "framework.cuh"

/**
 * @brief Copy current guest(GPU) memory to host(CPU)
 * 
 * @todo the outputSize should be stored (differ from OpenCL version)
 */
void framework::layer::detach()
{
#ifdef __DEBUG__
    printf("%s OutputSize : %ld\n", name.c_str(), outputSize);
#endif
    if(cMEM == nullptr)
        cMEM = new float[outputSize / sizeof(float)];

    hipError_t error_id;

	error_id = hipMemcpy(cMEM, gMEM, outputSize, hipMemcpyDeviceToHost);
	if (error_id != hipSuccess) {
		printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipDeviceSynchronize();
	if (error_id != hipSuccess) {
		printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
}

