#include "hip/hip_runtime.h"
#include "framework.cuh"

/**
 * @brief Batch Oriented 2d Convolutional layer.
 * 
 * @param inputH Input map height
 * @param inputW Input map width
 * @param cin Input channel size
 * @param outputH Output map height
 * @param outputW Output map width
 * @param cout Output channel size
 * @param batch Batch Size
 * @param stride Convolutional layer stride
 * @param kSize Convolutional layer kernel size
 * @param pad Convolutional padding size
 * @param X Input buffer pointer
 * @param W Weight buffer pointer
 * @param B Bias buffer pointer
 * @param Y Output buffer pointer
 */
__global__ void Conv2D_cout(int inputH, int inputW, int cin, int outputH, int outputW, int cout, int batchSize, int stride, int kSize, int pad, float* X, float* W, float* B, float* Y)
{
    unsigned int x_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_global_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z_global_idx = blockIdx.z * blockDim.z + threadIdx.z;

#ifdef __KERNEL_DEBUG__
    if ((x_global_idx + y_global_idx + z_global_idx) == 0) {
        unsigned int x_global_size = gridDim.x * blockDim.x;
        unsigned int y_global_size = gridDim.y * blockDim.y;
        unsigned int z_global_size = gridDim.z * blockDim.z;
        
        printf("Grid Size : [%u, %u, %u]\n", gridDim.x, gridDim.y, gridDim.z);
        printf("Block Size : [%u, %u, %u]\n", blockDim.x, blockDim.y, blockDim.z);
        printf("Global Size : [%u, %u, %u]\n", x_global_size, y_global_size, z_global_size);
        printf("Input Height : %d\n", inputH);
        printf("Input Width : %d\n", inputW);
        printf("Input Channel : %d\n", cin);
        printf("Output Height : %d\n", outputH);
        printf("Output Width : %d\n", outputW);
        printf("Output Channel : %d\n", cout);
        printf("Batch Size : %d\n", batchSize);
        printf("Stride : %d\n", stride);
        printf("Kernel Size : %d\n", kSize);
        printf("Padding : %d\n", pad);
    }
#endif

    int width  = x_global_idx;
    int height = y_global_idx;
    int cOut   = z_global_idx;

    int wSubSize = kSize * kSize * cin;
    int xOneBatchSize = inputH * inputW * cin;
    int xMapSize = inputH * inputW;
    int yOneBatchSize = outputH * outputW * cout;
    int yMapSize = outputH * outputW;

    float  sum;
    int indexW;
    for (int N = 0; N < batchSize; ++N)
    {
        sum    = 0;
        indexW = 0; // indexW = cIn * kh * kw + kh * kw + kw
        for (int cIn = 0; cIn < cin; ++cIn)
        {
            for(int kh = 0; kh < kSize; ++kh)
            {
                for (int kw = 0; kw < kSize; ++kw, ++indexW)
                {
                    // Consider the padded space has at least one coordinate out of bound e.g.[0, inputH] (top left corner start a [-1, -1])
                    
                    int hp = height * stride + kh - pad;
                    int wp = width  * stride + kw - pad;
                    if(hp >= 0 && wp >= 0 && hp < inputH && wp < inputW)
                        sum += W[cOut * wSubSize + indexW] * X[N * xOneBatchSize + cIn * xMapSize + hp * inputW + wp];
                }
            }
        }
        if(B != nullptr) {
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum + B[cOut];
        }
        else {
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum;
        }
    }
}


__global__ void Conv2D_shared(int inputH, int inputW, int cin, int outputH, int outputW, int cout, int batchSize, int stride, int kSize, int pad, float* X, float* W, float* B, float* Y)
{
    unsigned int x_global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_global_idx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z_global_idx = blockIdx.z * blockDim.z + threadIdx.z;

#ifdef __KERNEL_DEBUG__
    if ((x_global_idx + y_global_idx + z_global_idx) == 0) {
        unsigned int x_global_size = gridDim.x * blockDim.x;
        unsigned int y_global_size = gridDim.y * blockDim.y;
        unsigned int z_global_size = gridDim.z * blockDim.z;
        
        printf("Grid Size : [%u, %u, %u]\n", gridDim.x, gridDim.y, gridDim.z);
        printf("Block Size : [%u, %u, %u]\n", blockDim.x, blockDim.y, blockDim.z);
        printf("Global Size : [%u, %u, %u]\n", x_global_size, y_global_size, z_global_size);
        printf("Input Height : %d\n", inputH);
        printf("Input Width : %d\n", inputW);
        printf("Input Channel : %d\n", cin);
        printf("Output Height : %d\n", outputH);
        printf("Output Width : %d\n", outputW);
        printf("Output Channel : %d\n", cout);
        printf("Batch Size : %d\n", batchSize);
        printf("Stride : %d\n", stride);
        printf("Kernel Size : %d\n", kSize);
        printf("Padding : %d\n", pad);
    }
#endif

    int width  = x_global_idx;
    int height = y_global_idx;
    int cOut   = z_global_idx;
    
    int wSubSize = kSize * kSize * cin;
    int xOneBatchSize = inputH * inputW * cin;
    int xMapSize = inputH * inputW;
    int yOneBatchSize = outputH * outputW * cout;
    int yMapSize = outputH * outputW;

    // Setup shared memory
    extern __shared__ float sharedWeight[];

    int indexW = 0;
#ifdef DEBUG_OLD_SHARE
    for (; indexW < wSubSize; ++indexW)
        sharedWeight[threadIdx.z * wSubSize + indexW] = W[cOut * wSubSize + indexW];
#endif

    int sharedNum = wSubSize * blockDim.z;
    int threadLocalIdx = threadIdx.z * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x;
    int threadLocalDim = blockDim.x * blockDim.y * blockDim.z;

    // TODO : Might be a better solution
    /*
    for (int i = 0; i <= sharedNum / threadLocalDim; ++i) {
        if ((i * threadLocalDim + threadLocalIdx) < sharedNum)
            sharedWeight[i * threadLocalDim + threadLocalIdx] = W[(sharedNum * blockIdx.z) + (i * threadLocalDim) + threadLocalIdx];
    }
    */

    for (int i = 0; i < sharedNum ; i += threadLocalDim) {
        if ((i + threadLocalIdx) < sharedNum)
            sharedWeight[i + threadLocalIdx] = W[(sharedNum * blockIdx.z) + i + threadLocalIdx];
    }
    __syncthreads();
    
    float sum;
    for (int N = 0; N < batchSize; ++N)
    {
        sum    = 0;
        indexW = 0; // indexW = cIn * kh * kw + kh * kw + kw
        for (int cIn = 0; cIn < cin; ++cIn)
        {
            for(int kh = 0; kh < kSize; ++kh)
            {
                for (int kw = 0; kw < kSize; ++kw, ++indexW)
                {
                    int hp = height * stride + kh - pad;
                    int wp = width  * stride + kw - pad;
                    if(hp >= 0 && wp >= 0 && hp < inputH && wp < inputW)
                        sum += sharedWeight[threadIdx.z * wSubSize + indexW] * X[N * xOneBatchSize + cIn * xMapSize + hp * inputW + wp];
                }
            }
        }
        if(B != nullptr) {
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum + B[cOut];
        }
        else {
            Y[N * yOneBatchSize + cOut * yMapSize + height * outputW + width] = sum;
        }
    }
}

framework::conv2d::conv2d(layer* p, int cout, int ks, int stride, int pad, bool use_bias, uint8_t version, dim3 blockSize)
{
    name = "Conv2D";
    useBias = use_bias;
    padSize = pad;
    this->stride = stride>2?2:stride;
    kSize = ks;
    prvLayer = p;
    prvShape = prvLayer->getShape();
    shape.N = prvShape.N;
    shape.C = cout;
    shape.H = (prvShape.H - kSize + 2 * padSize) / stride + 1;
    shape.W = (prvShape.W - kSize + 2 * padSize) / stride + 1;
    memSize = shape.N * shape.C * shape.H * shape.W * sizeof(float);
    
    // Select compute version (NoShared : 0, WithShared : 1)
    this->version = version;

    hipError_t error_id;
    
    error_id = hipMalloc(&gMEM, memSize);
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

    error_id = hipDeviceSynchronize();
    if (error_id != hipSuccess) {
        printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    weightSize = shape.C * prvShape.C * kSize * kSize;
    biasSize = shape.C;
    setNParam(weightSize + int(useBias) * biasSize);
    outputSize = shape.N * shape.C * shape.H * shape.W * sizeof(float);

    dimBlock.x = blockSize.x;
    dimBlock.y = blockSize.y;
    dimBlock.z = blockSize.z;

    dimGrid.x = (shape.W + blockSize.x - 1) / blockSize.x;
    dimGrid.y = (shape.H + blockSize.y - 1) / blockSize.y;
    dimGrid.z = (shape.C + blockSize.z - 1) / blockSize.z;

    jobSize.x = shape.W;
    jobSize.y = shape.H;
    jobSize.z = shape.C;
#ifdef __DEBUG__
    printf("%s\nGlobal Work Size [%u, %u, %u]\nGPU Memory Size : %ld\n", name.c_str(), (dimGrid.x * dimBlock.x), (dimGrid.y * dimBlock.y), (dimGrid.z * dimBlock.z), memSize);
    printf("\nBlockDim : [%u, %u, %u]\nGridDim : [%u, %u, %u]\n", dimBlock.x, dimBlock.y, dimBlock.z, dimGrid.x, dimGrid.y, dimGrid.z);
#endif
}

/**
 * @brief Launch Kernel with Hyperparameter set in constructor.
 * 
 */
void framework::conv2d::run()
{
	hipError_t error_id;
    float* prvMEM = prvLayer->getGPUMem();

    switch (version) {
        case 0:
	        Conv2D_cout<<<dimGrid, dimBlock>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
            break;
        case 1:
            int sharedMem = kSize * kSize * prvShape.C * dimBlock.z * sizeof(float);
#ifdef __DEBUG__
            printf("%s Shared Memory Size : %d Shared Memory Num : %d\n", name.c_str(), sharedMem, sharedNum);
#endif
            Conv2D_shared<<<dimGrid, dimBlock, sharedMem>>>(prvShape.H, prvShape.W, prvShape.C, shape.H, shape.W, shape.C, shape.N, stride, kSize, padSize, prvMEM, wMEM, bMEM, gMEM);
            break;
    }
	hipDeviceSynchronize();
    error_id = hipGetLastError();
    if (error_id != hipSuccess) {
		printf("Error: %d\n%s\n\n", static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
}

/**
 * @brief Set layer weight & bias parameter
 * 
 * @param buffer CPU buffer to initial weight & bias parameter
 */
size_t framework::conv2d::setParam(float* buffer)
{
  	hipError_t error_id;

    error_id = hipMalloc(&wMEM, weightSize * sizeof(float));
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipMemcpy(wMEM, buffer, weightSize * sizeof(float), hipMemcpyHostToDevice);
	if (error_id != hipSuccess) {
		printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

	error_id = hipDeviceSynchronize();
	if (error_id != hipSuccess) {
		printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}
    
    if (useBias) {
        error_id = hipMalloc(&bMEM, biasSize * sizeof(float));
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipMemcpy(bMEM, (buffer + weightSize), biasSize * sizeof(float), hipMemcpyHostToDevice);
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipDeviceSynchronize();
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }
    }
    return weightSize + int(useBias) * biasSize;
}

