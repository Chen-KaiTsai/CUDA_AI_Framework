#include "framework.cuh"

framework::input::input(shape_t s)
{
    name = "Input";
    prvLayer = nullptr;
    shape = s;
    memSize = s.N * s.C * s.H * s.W * sizeof(float);

    hipError_t error_id;
    
    error_id = hipMalloc(&gMEM, memSize);
	if (error_id != hipSuccess) {
		printf("Error %s hipMalloc() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
		exit(EXIT_FAILURE);
	}

    error_id = hipDeviceSynchronize();
    if (error_id != hipSuccess) {
        printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }
    outputSize = s.N * s.C * s.H * s.W * sizeof(float);
#ifdef __DEBUG__
    printf("%s\nGPU Memory Size : %ld\n", name.c_str(), memSize);
#endif
}

void framework::input::run()
{
    if (cMEM != nullptr) {
        hipError_t error_id;

	    error_id = hipMemcpy(gMEM, cMEM, memSize, hipMemcpyHostToDevice);
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipMemcpy() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }

	    error_id = hipDeviceSynchronize();
	    if (error_id != hipSuccess) {
	    	printf("Error %s hipDeviceSynchronize() : %d\n%s\n\n", name.c_str(), static_cast<int>(error_id), hipGetErrorString(error_id));
	    	exit(EXIT_FAILURE);
	    }
    }
}

size_t framework::input::setParam(float* buffer) { return 0; };
